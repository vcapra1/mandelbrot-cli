#include "hip/hip_runtime.h"
#include "compute.cuh"

#include <stdio.h>
#include <string.h>
#include <unistd.h>

__device__ Real mul(Real a, Real b) { return a * b; }
__device__ Real add(Real a, Real b) { return a + b; }
__device__ Real sub(Real a, Real b) { return a - b; }
__device__ Real mag_sq(Complex c) { return add(mul(c.real, c.real), mul(c.imag, c.imag)); }

__device__ Complex f(Complex z, Complex c) {
	Real real = add(sub(mul(z.real, z.real), mul(z.imag, z.imag)), c.real);
	Real w = mul(z.real, z.imag);
	Real imag = add(add(w, w), c.imag);

	return { real, imag };
}

__global__ void compute(Pixel *pixels, 
						unsigned long width, 
						unsigned long height, 
						unsigned long iterations, 
						unsigned long long *progress) {

	// Figure out which pixel this thread is responsible for
	unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < width) {
		unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
		if (y < height) {
			unsigned long idx = x + y * width;

			// Get a pointer to our pixel
			Pixel *pixel = &pixels[idx];

			// Loop until the pixel diverges, or the max iterations is reached
			while (pixel->i < iterations && !pixel->d) {
				pixel->z = f(pixel->z, pixel->c);
				pixel->i += 1;

				// Check to see if it's diverged
				if (mag_sq(pixel->z) > 4.0) {
					pixel->d = true;
				}
			}

			// Increment the progress
			atomicAdd(progress, 1);
		}
	}
}

extern "C" {
	uint32_t cuda_compute(uint32_t iterations, RenderData data, void **progress) {
		// Make sure the image isn't too big
		if (data.width > 2097120 || data.height > 2097120) {
			// Too big :( TODO: not really, we can go quite a bit bigger, but we'll do that later
			return 99999;
		}

		// Keep track of errors
		hipError_t status = hipSuccess;

		// Allocate managed memory for the progress
		unsigned long long *progress_shared;
		status = hipMallocManaged((void **)&progress_shared, sizeof(unsigned long long));

		if (status != hipSuccess) { return status; }

		// Set the progress to zero
		*progress_shared = 0;

		// Pass the reference to the progress back through the double pointer
		*progress = (void *)progress_shared;

		// Allocate managed memory for the pixels
		Pixel *pixels;
		unsigned int data_length = sizeof(Pixel) * data.num;
		status = hipMallocManaged((void **)&pixels, data_length);

		if (status != hipSuccess) { return status; }

		// Copy pixels data from host to managed memory
		memcpy(pixels, data.pixels, data_length);

		// Calculate grid and block sizes
		dim3 blockSize(32, 32, 1);
		dim3 gridSize(data.width / blockSize.x, data.height / blockSize.y, 1);

		// Round up
		if (data.width % blockSize.x) { gridSize.x += 1; }
		if (data.height % blockSize.y) { gridSize.y += 1; }

		// Run kernel
		compute<<<gridSize, blockSize>>>(pixels, data.width, data.height, iterations, progress_shared);
		status = hipPeekAtLastError();

		if (status != hipSuccess) { return status; }

		status = hipDeviceSynchronize();

		if (status != hipSuccess) { return status; }

		// Copy pixels data back to original memory
		memcpy(data.pixels, pixels, data_length);

		// Clear the progress reference
		*progress = 0;
		
		// Wait to make sure the progress thread (in Rust) doesn't try to access freed memory
		sleep(1);

		// Free memory
		hipFree(progress_shared);
		progress_shared = 0;
		
		hipFree(pixels);
		pixels = 0;

		return 0;
	}
}

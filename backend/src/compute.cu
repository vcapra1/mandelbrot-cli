#include "hip/hip_runtime.h"
#include "compute.cuh"

#include <stdio.h>
#include <string.h>
#include <unistd.h>

__global__ void compute(Pixel *pixels, unsigned long width, unsigned long height) {
	// Figure out which pixel this thread is responsible for
	unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < width) {
		unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
		if (y < height) {
			unsigned long idx = x + y * width;

			// Get a pointer to our pixel
			Pixel *pixel = &pixels[idx];

			// TODO
		}
	}
}

extern "C" {
	uint32_t cuda_compute(uint32_t iterations, RenderData data, uint32_t *progress) {
		// Print information to ensure it was all transferred properly
		//printf("Iterations: %u\n", iterations);
		//printf("RenderData:\n");
		//printf("  Iterations: %u\n", data.iterations); 
		//printf("  Num: %u\n", data.num);
		//printf("  Pixels: %p\n", (void *)data.pixels);
		//for (unsigned int i = 0; i < data.num; i++) {
		//	if (i == (unsigned int) 7 || i == (unsigned int) (data.num / 2 + 100)) {
		//		printf("    ...\n");
		//	}
		//	if (i >= (unsigned int) 7 && i < (unsigned int) (data.num / 2 + 97)) {
		//		continue;
		//	}
		//	if (i >= (unsigned int) (data.num / 2 + 100) && i < (unsigned int) (data.num - 3)) {
		//		continue;
		//	}

		//	printf("    %d: (i: %d, d: %s, c: (%f, %f), z: (%f, %f))\n", i, data.pixels[i].i,
		//		data.pixels[i].d ? "true" : "false", data.pixels[i].c.real, data.pixels[i].c.imag,
		//		data.pixels[i].z.real, data.pixels[i].z.imag);
		//}

		if (data.width > 2097120 || data.height > 2097120) {
			// Too big :(
			return 99999;
		}

		// Keep track of errors
		hipError_t status = hipSuccess;

		// Allocate managed memory for the pixels
		Pixel *pixels;
		unsigned int data_length = sizeof(Pixel) * data.num;
		status = hipMallocManaged((void **)&pixels, data_length);

		if (status != hipSuccess) { return status; }

		// Copy pixels data from host to managed memory
		memcpy(pixels, data.pixels, data_length);

		// Calculate grid and block sizes
		dim3 blockSize(32, 32, 1);
		dim3 gridSize(data.width / blockSize.x, data.height / blockSize.y, 1);

		// Round up
		if (data.width % blockSize.x) { gridSize.x += 1; }
		if (data.height % blockSize.y) { gridSize.y += 1; }

		// Run kernel
		compute<<<gridSize, blockSize>>>(pixels, data.width, data.height);
		status = hipPeekAtLastError();
		
		if (status != hipSuccess) { return status; }

		status = hipDeviceSynchronize();

		if (status != hipSuccess) { return status; }

		// Copy pixels data back to original memory
		memcpy(data.pixels, pixels, data_length);

		// Free memory
		hipFree(pixels);

		return 0;
	}
}

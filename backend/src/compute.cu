#include "hip/hip_runtime.h"
#include "compute.cuh"

#include <stdio.h>
#include <string.h>
#include <unistd.h>

__device__ Real mul(Real a, Real b) { return a * b; }
__device__ Real add(Real a, Real b) { return a + b; }
__device__ Real sub(Real a, Real b) { return a - b; }
__device__ Real mag_sq(Complex c) { return add(mul(c.real, c.real), mul(c.imag, c.imag)); }

__device__ Complex f(Complex z, Complex c) {
	Real real = add(sub(mul(z.real, z.real), mul(z.imag, z.imag)), c.real);
	Real w = mul(z.real, z.imag);
	Real imag = add(add(w, w), c.imag);

	return { real, imag };
}

__global__ void compute(Pixel *pixels, unsigned long width, unsigned long height, unsigned long iterations) {
	// Figure out which pixel this thread is responsible for
	unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < width) {
		unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
		if (y < height) {
			unsigned long idx = x + y * width;

			// Get a pointer to our pixel
			Pixel *pixel = &pixels[idx];

			// Loop until the pixel diverges, or the max iterations is reached
			while (pixel->i < iterations && !pixel->d) {
				pixel->z = f(pixel->z, pixel->c);
				pixel->i += 1;
				
				// Check to see if it's diverged
				if (mag_sq(pixel->z) > 4.0) {
					pixel->d = true;
				}
			}
		}
	}
}

extern "C" {
	uint32_t cuda_compute(uint32_t iterations, RenderData data, uint32_t *progress) {
		// Print information to ensure it was all transferred properly
		//printf("Iterations: %u\n", iterations);
		//printf("RenderData:\n");
		//printf("  Iterations: %u\n", data.iterations); 
		//printf("  Num: %u\n", data.num);
		//printf("  Pixels: %p\n", (void *)data.pixels);
		//for (unsigned int i = 0; i < data.num; i++) {
		//	if (i == (unsigned int) 7 || i == (unsigned int) (data.num / 2 + 100)) {
		//		printf("    ...\n");
		//	}
		//	if (i >= (unsigned int) 7 && i < (unsigned int) (data.num / 2 + 97)) {
		//		continue;
		//	}
		//	if (i >= (unsigned int) (data.num / 2 + 100) && i < (unsigned int) (data.num - 3)) {
		//		continue;
		//	}

		//	printf("    %d: (i: %d, d: %s, c: (%f, %f), z: (%f, %f))\n", i, data.pixels[i].i,
		//		data.pixels[i].d ? "true" : "false", data.pixels[i].c.real, data.pixels[i].c.imag,
		//		data.pixels[i].z.real, data.pixels[i].z.imag);
		//}

		if (data.width > 2097120 || data.height > 2097120) {
			// Too big :( TODO: not really, we can go quite a bit bigger, but we'll do that later
			return 99999;
		}

		// Keep track of errors
		hipError_t status = hipSuccess;

		// Allocate managed memory for the pixels
		Pixel *pixels;
		unsigned int data_length = sizeof(Pixel) * data.num;
		status = hipMallocManaged((void **)&pixels, data_length);

		if (status != hipSuccess) { return status; }

		// Copy pixels data from host to managed memory
		memcpy(pixels, data.pixels, data_length);

		// Calculate grid and block sizes
		dim3 blockSize(32, 32, 1);
		dim3 gridSize(data.width / blockSize.x, data.height / blockSize.y, 1);

		// Round up
		if (data.width % blockSize.x) { gridSize.x += 1; }
		if (data.height % blockSize.y) { gridSize.y += 1; }

		// Run kernel
		compute<<<gridSize, blockSize>>>(pixels, data.width, data.height, iterations);
		status = hipPeekAtLastError();
		
		if (status != hipSuccess) { return status; }

		status = hipDeviceSynchronize();

		if (status != hipSuccess) { return status; }

		// Copy pixels data back to original memory
		memcpy(data.pixels, pixels, data_length);

		// Free memory
		hipFree(pixels);

		// Set progress to 100%
		*progress = 100;

		return 0;
	}
}
